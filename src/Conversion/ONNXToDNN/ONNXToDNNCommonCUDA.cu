//===------ Common CUDA functions for ONNXToDNN Pass ------===//

#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <iostream>

#include "src/Conversion/ONNXToDNN/ONNXToDNNCommonCUDA.cuh"

int64_t calculateWorkspace(
    int64_t dimX[4], int64_t dimw[4],
    int64_t pads[2], int64_t strides[2],
    int64_t convAlgorithm, int64_t group) {

  hipdnnHandle_t cudnnHandle;
  hipdnnTensorDescriptor_t inTensorDesc, outTensorDesc;
  hipdnnFilterDescriptor_t filterDesc;
  hipdnnConvolutionDescriptor_t convDesc;

  hipdnnCreate(&cudnnHandle);
  hipdnnCreateTensorDescriptor(&inTensorDesc);
  hipdnnCreateTensorDescriptor(&outTensorDesc);
  hipdnnCreateFilterDescriptor(&filterDesc);
  hipdnnCreateConvolutionDescriptor(&convDesc);

  hipdnnSetTensor4dDescriptor(
      inTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
      dimX[0], dimX[1], dimX[2], dimX[3]);

  hipdnnSetFilter4dDescriptor(
      filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
      dimw[0], dimw[1], dimw[2], dimw[3]);


  hipdnnSetConvolution2dDescriptor(convDesc,
      (int)pads[0], (int)pads[1],
      (int)strides[0], (int)strides[1],
      1, 1,
      HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);
  hipdnnSetConvolutionGroupCount(convDesc, (int)group);

  int out_n, out_c, out_h, out_w;
  hipdnnGetConvolution2dForwardOutputDim(convDesc, inTensorDesc, filterDesc, &out_n, &out_c, &out_h, &out_w);
  hipdnnSetTensor4dDescriptor(outTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w);

  size_t sizeInBytes = 0;
  hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
      inTensorDesc,
      filterDesc,
      convDesc,
      outTensorDesc,
      (hipdnnConvolutionFwdAlgo_t) convAlgorithm,
      &sizeInBytes);

  return sizeInBytes;
}

int64_t calculateConvAlgo(
    int64_t dimX[4], int64_t dimw[4],
    int64_t pads[4], int64_t strides[4], int64_t group) {

  hipdnnHandle_t cudnnHandle;
  hipdnnTensorDescriptor_t inTensorDesc, outTensorDesc;
  hipdnnFilterDescriptor_t filterDesc;
  hipdnnConvolutionDescriptor_t convDesc;

  hipdnnCreate(&cudnnHandle);
  hipdnnCreateTensorDescriptor(&inTensorDesc);
  hipdnnCreateTensorDescriptor(&outTensorDesc);
  hipdnnCreateFilterDescriptor(&filterDesc);
  hipdnnCreateConvolutionDescriptor(&convDesc);

  hipdnnSetTensor4dDescriptor(
      inTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
      dimX[0], dimX[1], dimX[2], dimX[3]);

  hipdnnSetFilter4dDescriptor(
      filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
      dimw[0], dimw[1], dimw[2], dimw[3]);


  hipdnnSetConvolution2dDescriptor(convDesc,
      (int)pads[0], (int)pads[1],
      (int)strides[0], (int)strides[1],
      1, 1,
      HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

  hipdnnSetConvolutionGroupCount(convDesc, (int)group);

  int out_n, out_c, out_h, out_w;
  hipdnnGetConvolution2dForwardOutputDim(convDesc, inTensorDesc, filterDesc, &out_n, &out_c, &out_h, &out_w);
  hipdnnSetTensor4dDescriptor(outTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w);

  int returnedAlgoNum = -1;
  hipdnnConvolutionFwdAlgoPerf_t algoPerf[7];

  hipdnnFindConvolutionForwardAlgorithm(cudnnHandle,
      inTensorDesc,
      filterDesc,
      convDesc,
      outTensorDesc,
      7,
      &returnedAlgoNum,
      &algoPerf[0]);
  if (((int64_t) algoPerf[0].algo == 0) || ((int64_t) algoPerf[1].algo == 0) ||
      ((int64_t) algoPerf[2].algo == 0) || ((int64_t) algoPerf[3].algo == 0) ||
      ((int64_t) algoPerf[4].algo == 0) || ((int64_t) algoPerf[5].algo == 0) ||
      ((int64_t) algoPerf[6].algo == 0)) {
    return (int64_t) 0;
  } else {
    return (int64_t)algoPerf[0].algo;
  }
}


