#include <stdio.h>
#include <hip/hip_runtime.h>

#define DEBUG 0

//CUDA Kernel for ONNX Gather function (== numpy.take with axis data)
//covers inputRank <= 4, indicesRank <= 4
//which in result covers outputRank <= 7
__global__ void tensorGatherI64 (
    int64_t* X, int64_t* dimInput, int64_t inputDim, 
    int64_t* indices, int64_t* dimIndices, int64_t indicesDim, 
    int64_t axis, int64_t* Y, 
    int guard) {
  int64_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<guard){
    int64_t in_str[4]={0,};
    in_str[0]=1;
    for(int j=1; j<inputDim; j++){
      in_str[j]=in_str[j-1]*dimInput[inputDim-j];
    }

    int64_t ind_str[4];
    ind_str[0]=1;
    for(int j=1; j<indicesDim; j++){
      ind_str[j]=ind_str[j-1]*dimIndices[indicesDim-j];
    }
    int64_t ind_size=1;
    for(int j=0; j<indicesDim; j++){
      ind_size*=dimIndices[j];
    }

    int64_t dimOutput[4];
    for(int j=0; j<inputDim; j++){
      dimOutput[j]=dimInput[j];
    }
    dimOutput[axis]=ind_size;

    int64_t out_str[4];
    out_str[0]=1;
    for(int j=1; j<inputDim; j++){
      out_str[j]=out_str[j-1]*dimOutput[inputDim-j];
    }
    int64_t out_index[4];
    out_index[0]=i/out_str[inputDim-1];
    for(int j=1; j<inputDim; j++){
      out_index[j]=(i%out_str[inputDim-j])/out_str[inputDim-1-j];
    }

    int64_t input_index[4];
    for(int j=0; j<inputDim; j++){
      input_index[j]=out_index[j];
    }
    input_index[axis]=indices[out_index[axis]];

    int64_t input_index_val=0;
    for(int j=0; j<inputDim; j++){
      input_index_val+=input_index[j]*in_str[inputDim-1-j];
    }
    Y[i] = X[input_index_val];
  }
}

extern "C"
int64_t* CUDAGatherInt64Func (
    int64_t* inData_d, int64_t* dimInput, int64_t inputRank,
    int64_t* outData_d, int64_t* dimOutput, int64_t outputRank,
    int64_t* indicesData_d, int64_t* dimIndices, int64_t indicesRank,
    int64_t axis) {

  int64_t* dimInput_d;
  hipMalloc((void**)&dimInput_d, sizeof(int64_t*)*inputRank);
  hipMemcpy(dimInput_d, dimInput, sizeof(int64_t*)*inputRank, hipMemcpyHostToDevice);

  int64_t* dimIndices_d;
  hipMalloc((void**)&dimIndices_d, sizeof(int64_t*)*indicesRank);
  hipMemcpy(dimIndices_d, dimIndices, sizeof(int64_t*)*indicesRank, hipMemcpyHostToDevice);

  int64_t guard=1;
  for(int i=0; i<outputRank; i++) guard*=dimOutput[i];
  int64_t numCTA = (guard+1024-1)/1024;

  tensorGatherI64 <<<numCTA, 1024>>> (inData_d, dimInput_d, inputRank, indicesData_d, dimIndices_d, indicesRank, axis, outData_d, guard);

#if DEBUG
  int sizeA = 1;
  for(int i=0; i<inputRank; i++) sizeA*=dimInput[i];

  float *X;
  X = (float*) malloc(sizeof(float) * sizeA);
  cudaMemcpy(X, inData_d, sizeof(float) * sizeA, (cudaMemcpyKind) 2);

  printf("\n[Gather] tensor size : %d", sizeA);

  if(sizeA >= 8)
    printf("\n[Gather] inData_d -> %.5f, %.5f, %.5f, %.5f, %.5f, %.5f, %.5f, %.5f\n",
        X[0], X[1], X[2], X[3], X[4], X[5], X[6], X[7]);
  else
    printf("[Gather] inData_d -> %.5f\n", X[0]);

  float *Z;
  Z = (float*) malloc(sizeof(float) * guard);
  cudaMemcpy(Z, outData_d, sizeof(float) * guard, (cudaMemcpyKind) 2);

  printf("[Gather] outData_d Addr -> %p\n", outData_d);
  printf("[Gather] outData_d -> %.5f, %.5f, %.5f, %.5f, %.5f, %.5f, %.5f, %.5f\n\n\n",
      Z[0], Z[1], Z[2], Z[3], Z[4], Z[5], Z[6], Z[7]);
  free(Z);
#endif


  return outData_d;
}
