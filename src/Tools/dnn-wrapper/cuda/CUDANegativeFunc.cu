#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void tensorNegative (float* inData_d, float* outData_d, int64_t guard) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i<guard) {
    outData_d[i] = -(inData_d[i]);
  }

}

extern "C"
float* CUDANegativeFunc (
    float* inData_d, float* outData_d,
    int64_t* dimInput, int64_t rank) {

  int64_t guard = 1;
  for (int i=0; i<rank; i++) {
    guard *= dimInput[i];
  }
  int64_t numCTA = (guard+1024-1)/1024;

  tensorNegative <<<numCTA, 1024>>> (
      inData_d, outData_d, guard);

  return outData_d;
}
