#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void tensorTranspose4DInt64 (
    int64_t* inData_d, int64_t* dimInput,
    int64_t* outData_d, int64_t* perm, int64_t guard) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i<guard) {
    int64_t str0 = 1;
    int64_t str1 = dimInput[3];
    int64_t str2 = dimInput[3] * dimInput[2];
    int64_t str3 = dimInput[3] * dimInput[2] * dimInput[1];

    int64_t indices[4] = {(i/str3), ((i%str3)/str2), ((i%str2)/str1), (i%str1)};
    int64_t outputShape[4] = {dimInput[perm[0]],dimInput[perm[1]],
      dimInput[perm[2]],dimInput[perm[3]]};

    int64_t outstr0 = 1;
    int64_t outstr1 = outputShape[3];
    int64_t outstr2 = outputShape[3] * outputShape[2];
    int64_t outstr3 = outputShape[3] * outputShape[2] * outputShape[1];

    outData_d [indices[perm[0]] * outstr3 +
              indices[perm[1]] * outstr2 +
              indices[perm[2]] * outstr1 +
              indices[perm[3]] * outstr0] = inData_d [indices[0] * str3 +
                                                      indices[1] * str2 +
                                                      indices[2] * str1 +
                                                      indices[3] * str0];
  }
}

extern "C"
int64_t* CUDATranspose4DInt64Func (
    int64_t* inData_d, int64_t* dimInput,
    int64_t*  outData_d,  int64_t* dimOutput,
    int64_t* perm, int64_t rank) {

  int64_t guard = 1;
  for (int i=0; i<rank; i++) {
    guard *= dimOutput[i];
  }
  int64_t numCTA = (guard+1024-1)/1024;

  int64_t* perm_d;
  hipMalloc((void**)&perm_d, sizeof(int64_t)*4);
  hipMemcpy(perm_d, perm, sizeof(int64_t)*4, hipMemcpyHostToDevice);

  int64_t* inShape_d;
  hipMalloc((void**)&inShape_d, sizeof(int64_t)*4);
  hipMemcpy(inShape_d, dimInput, sizeof(int64_t)*4, hipMemcpyHostToDevice);

  tensorTranspose4DInt64 <<<numCTA, 1024>>> (
      inData_d, inShape_d, outData_d, perm_d, guard);

  return outData_d;
}
