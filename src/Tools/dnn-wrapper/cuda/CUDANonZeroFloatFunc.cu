#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void tensorNonZeroF32 (
    float* inData_d, float* outData_d, int64_t num) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i == 0) {
    int currIdx = 0;
    for (int j = 0; j < num; j++) {
      if(inData_d[j] != 0.f) {
        outData_d[currIdx] = (float)j;
        currIdx++;
      }
    }
  }
}

extern "C"
float* CUDANonZeroFloatFunc (
    float* inData_d, int64_t* dimInput,
    float*  outData_d, int64_t* dimOutput, int64_t rank) {

  int64_t elemNum = 1;
  for (int64_t i=0; i<rank; i++) {
    elemNum *= dimInput[i];
  }

  tensorNonZeroF32 <<<1, 1>>> (
      inData_d, outData_d, elemNum);

  return outData_d;
}
