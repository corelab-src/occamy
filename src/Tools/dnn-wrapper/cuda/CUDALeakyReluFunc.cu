#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void tensorLeakyRelu (
    float* inData_d, float* outData_d,
    float alpha, int64_t guard) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i<guard) {
    if(inData_d[i] < 0)
      outData_d[i] = alpha * inData_d[i];
    else 
      outData_d[i] = inData_d[i];
  }
}

extern "C"
float* CUDALeakyReluFunc (
    float* inData_d,
    float* outData_d, int64_t* dimOutput,
    float alpha, int64_t rank) {

  int64_t guard = 1;
  for (int i=0; i<rank; i++) {
    guard *= dimOutput[i];
  }
  int64_t numCTA = (guard+1024-1)/1024;


  tensorLeakyRelu <<<numCTA, 1024>>> (
     inData_d, outData_d, alpha, guard);

  return outData_d;
}
