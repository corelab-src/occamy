#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblaslt.h>
#include <stdio.h>

#define DEBUG 0

extern "C"
float* CUBLASMatmul2dFunc(int64_t tensorDim,
    float *inDataA_d, int64_t dimInputA[2],
    float *inDataB_d, int64_t dimInputB[2],
    float *inDataC_d, int64_t dimInputC[2],
    float *outData_d, int64_t dimOutput[2],
    float alpha, float beta,
    int64_t transAAttr, int64_t transBAttr) {

#if DEBUG
  printf("\n[CUBLASMatmul2d] dimInputA -> %ld, %ld\n", dimInputA[0] , dimInputA[1]);
  printf("[CUBLASMatmul2d] dimInputB -> %ld, %ld\n", dimInputB[0] , dimInputB[1]);
  printf("[CUBLASMatmul2d] dimOutput -> %ld, %ld\n", dimOutput[0] , dimOutput[1]);

  float *X;
  X = (float*) malloc(sizeof(float) * dimInputA[0] * dimInputA[1]);
  cudaMemcpy(X, inDataA_d, sizeof(float) * dimInputA[0] * dimInputA[1], (cudaMemcpyKind) 2);

  float *X1;
  X1 = (float*) malloc(sizeof(float) * dimInputB[0] * dimInputB[1]);
  cudaMemcpy(X1, inDataB_d, sizeof(float) * dimInputB[0] * dimInputB[1], (cudaMemcpyKind) 2);

  float *X2;
  X2 = (float*) malloc(sizeof(float) * dimOutput[0] * dimOutput[1]);
  cudaMemcpy(X2, inDataC_d, sizeof(float) * dimOutput[0] * dimOutput[1], (cudaMemcpyKind) 2);

  printf("[CUBLASMatmul2d] inData_dA Addr -> %p, Size -> %lu\n", inDataA_d, sizeof(float) * dimInputA[0] * dimInputA[1]);
  printf("[CUBLASMatmul2d] inData_dA -> %.9f, %.9f, %.9f, %.9f, %.9f, %.9f, %.9f, %.9f\n\n",
      X[0], X[1], X[2], X[3], X[4], X[5], X[6], X[7]);

  printf("[CUBLASMatmul2d] inData_dB Addr -> %p, Size -> %lu\n", inDataB_d, sizeof(float) * dimInputB[0] * dimInputB[1]);
  printf("[CUBLASMatmul2d] inData_dB -> %.9f, %.9f, %.9f, %.9f, %.9f, %.9f, %.9f, %.9f\n\n",
      X1[0], X1[1], X1[2], X1[3], X1[4], X1[5], X1[6], X1[7]);

  printf("[CUBLASMatmul2d] inData_dC Addr -> %p, Size -> %lu\n", inDataC_d, sizeof(float) * dimOutput[0] * dimOutput[1]);
  printf("[CUBLASMatmul2d] inData_dC -> %.9f, %.9f, %.9f, %.9f, %.9f, %.9f, %.9f, %.9f\n\n",
      X2[0], X2[1], X2[2], X2[3], X2[4], X2[5], X2[6], X2[7]);

  free(X);
  free(X1);
  free(X2);
#endif

  hipblasOperation_t transA = transAAttr ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t transB = transBAttr ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  void* workspace;
  size_t workspaceSize = 1024 * 1024 * 4;
  hipStream_t stream;
  hipblasLtHandle_t ltHandle;

  hipblasLtCreate(&ltHandle);
  hipStreamCreate(&stream);
  hipMalloc(&workspace, workspaceSize);

  hipblasLtMatmulDesc_t operationDesc = NULL;
  hipblasLtMatrixLayout_t Adesc = NULL;
  hipblasLtMatrixLayout_t Bdesc = NULL;
  hipblasLtMatrixLayout_t Cdesc = NULL;
  hipblasLtMatrixLayout_t outdesc = NULL;
  hipblasLtMatmulPreference_t preference = NULL;

  int returnedResults = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};

  hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(transA));
  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(transB));

  hipblasLtOrder_t rowOrder = HIPBLASLT_ORDER_ROW;

  hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, dimInputA[0], dimInputA[1], dimInputA[1]);
  hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, (void*)&rowOrder, sizeof(rowOrder));

  hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, dimInputB[0], dimInputB[1], dimInputB[1]);
  hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, (void*)&rowOrder, sizeof(rowOrder));

  hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, dimOutput[0], dimOutput[1], dimOutput[1]);
  hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, (void*)&rowOrder, sizeof(rowOrder));

  hipblasLtMatrixLayoutCreate(&outdesc, HIP_R_32F, dimOutput[0], dimOutput[1], dimOutput[1]);
  hipblasLtMatrixLayoutSetAttribute(outdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, (void*)&rowOrder, sizeof(rowOrder));
#if DEBUG
  printf("%ld, %ld / %ld, %ld / %ld, %ld\n",
      dimInputA[0], dimInputA[1],
      dimInputB[0], dimInputB[1],
      dimOutput[0], dimOutput[1]
      );
#endif

  hipblasLtMatmulPreferenceCreate(&preference);
  hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));

  hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, outdesc, preference, 1, &heuristicResult, &returnedResults);

  if (returnedResults == 0) {
    printf("CUBLAS_STATUS_NOT_SUPPORTED\n");
  }
#if DEBUG
  else {
    printf("returnedResults : %d\n", returnedResults);
  }
#endif

  int matmulResult = (int)hipblasLtMatmul(ltHandle,
      operationDesc,
      &alpha,
      inDataA_d,
      Adesc,
      inDataB_d,
      Bdesc,
      &beta,
      outData_d,
      Cdesc,
      outData_d,
      Cdesc,
      &heuristicResult.algo,
      workspace,
      workspaceSize,
      stream);

#if DEBUG
  printf("\n[CUBLASMatmul2d] matmul result : %d\n" , matmulResult);
  float *y;
  y = (float*) malloc(sizeof(float) * dimOutput[0] * dimOutput[1]);
  cudaMemcpy(y, outData_d, sizeof(float) * dimOutput[0] * dimOutput[1], (cudaMemcpyKind) 2);

  printf("[CUBLASMatmul2d] outData_t Addr -> %p, Size -> %lu\n", outData_d, sizeof(float) * dimOutput[0] * dimOutput[1]);
  printf("[CUBLASMatmul2d] outData_t -> %.9f, %.9f, %.9f, %.9f, %.9f, %.9f, %.9f, %.9f\n\n",
      y[0], y[1], y[2], y[3], y[4], y[5], y[6], y[7]);
  free(y);
#endif

  // descriptors are no longer needed as all GPU work was already enqueued
  if (preference) hipblasLtMatmulPreferenceDestroy(preference);
  if (Cdesc) hipblasLtMatrixLayoutDestroy(Cdesc);
  if (Bdesc) hipblasLtMatrixLayoutDestroy(Bdesc);
  if (Adesc) hipblasLtMatrixLayoutDestroy(Adesc);
  if (operationDesc) hipblasLtMatmulDescDestroy(operationDesc);
  if (stream) hipStreamDestroy(stream);

  hipblasLtDestroy(ltHandle);
  hipFree(workspace);

  return outData_d;
}
